

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <pthread.h>

#define BLOCK_SIZE 64
#define REDUCTION_BLOCK_SIZE 1024
#define PIx2 6.2831853071795864769252867665590058f
#include <sys/time.h>

struct kValues {
	float Kx;
	float Ky;
	float Kz;
	float PhiMag;
};


//size needed: numK * 1
__global__ void ComputePhiMagGPU(struct kValues* kValsD, float* phiRD, float* phiID) {
	int indexK = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	//Shared memory is not needed since this is a coalesced access.
	//kVals.KxKyKz should be initialized in the host since it's pure memory operation. CUDA is not used for doing parrallel data memory operation.
	kValsD[indexK].PhiMag = phiRD[indexK] * phiRD[indexK] + phiID[indexK] * phiID[indexK];
}

__global__ void ImprovedReductionKernel(float* globalData, int interval, int dataSize) {
	int loc = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float data[REDUCTION_BLOCK_SIZE];
	if (loc * interval < dataSize) {
		//load to shared mem
		data[threadIdx.x] = globalData[loc];
		int stride = REDUCTION_BLOCK_SIZE / 2;
		do {
			__syncthreads();
			/*if (threadIdx.x == 0)
				printf("datasize=%d\n", dataSize);*/
			if (threadIdx.x < stride && threadIdx.x + stride < dataSize) {
				data[threadIdx.x] += data[threadIdx.x + stride];
				/*printf("%f,", data[threadIdx.x]);*/
			}
			stride >>= 1;
		} while (stride >= 1);
		if (threadIdx.x == 0) {
			globalData[loc] = data[0];
		}
	}
}

//size needed: numK * 1
__global__ void ComputeQGPU(float* globalqr, float* globalqi, struct kValues* globalkVals, float globalx, float globaly, float globalz) {
	//constant memory will limit the scalibility
	__shared__ float x, y, z;
	__shared__ struct kValues kVals[BLOCK_SIZE];
	__shared__ float Qracc[BLOCK_SIZE];
	__shared__ float Qiacc[BLOCK_SIZE];
	int indexK = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	//load shared mem
	kVals[threadIdx.x] = globalkVals[indexK];

	if (threadIdx.x == 0) {
		x = globalx;
		y = globaly;
		z = globalz;
	}
	__syncthreads();
	float expArg = PIx2 * (kVals[threadIdx.x].Kx * x + kVals[threadIdx.x].Ky * y + kVals[threadIdx.x].Kz * z);
	float cosArg, sinArg;
	sincosf(expArg, &sinArg, &cosArg);
	//the following should be zero for padding
	Qracc[threadIdx.x] = kVals[threadIdx.x].PhiMag * cosArg;
	Qiacc[threadIdx.x] = kVals[threadIdx.x].PhiMag * sinArg;

	//improved reduction
	int stride = BLOCK_SIZE / 2;
	do {
		__syncthreads();
		if (threadIdx.x < stride) {
			Qracc[threadIdx.x] += Qracc[threadIdx.x + stride];
			Qiacc[threadIdx.x] += Qiacc[threadIdx.x + stride];
		}
		stride >>= 1;
	} while (stride >= 1);
	if (threadIdx.x == 0) {
		*(globalqr + blockIdx.x) = Qracc[0];
		*(globalqi + blockIdx.x) = Qiacc[0];
	}
}

//cudaMalloc inside
void launchKernel(int numK, int numX, float* kxH, float* kyH, float* kzH, 
							float* xH, float* yH, float* zH, float* phiRH, float* phiIH, float* QrH, float* QiH,
							float** phiRD, float** phiID, struct kValues** kValsD) {
    struct timeval time0;
    struct timeval time1;
    struct timezone tz;
//    long kernelTime = 0;
//    long memoryTime = 0;
	//calculate dimension
	dim3 dim_grid, dim_block;
	dim_grid.x = numK / BLOCK_SIZE + (numK % BLOCK_SIZE == 0 ? 0 : 1);
	dim_grid.y = 1;
	dim_grid.z = 1;
	dim_block.x = BLOCK_SIZE;
	dim_block.y = 1;
	dim_block.z = 1;
	fflush(stdout);
	//prepare for calculating PhiMag
	hipMalloc(kValsD, dim_grid.x * BLOCK_SIZE * sizeof(struct kValues));
	struct kValues* kVals = (struct kValues*)calloc(numK, sizeof(struct kValues));
	for (int k = 0; k < numK; k++) {
		kVals[k].Kx = kxH[k];
		kVals[k].Ky = kyH[k];
		kVals[k].Kz = kzH[k];
	}
//    gettimeofday(&time0, &tz);
	hipMemset(*kValsD, 0, numK * sizeof(struct kValues));
	hipMemcpy(*kValsD, kVals, numK * sizeof(struct kValues), hipMemcpyHostToDevice);

	hipMalloc(phiRD, dim_grid.x * BLOCK_SIZE * sizeof(struct kValues));
	hipMemset(*phiRD, 0, numK * sizeof(float)); //0 * n = 0
	hipMemcpy(*phiRD, phiRH, numK * sizeof(struct kValues), hipMemcpyHostToDevice);

	hipMalloc(phiID, dim_grid.x * BLOCK_SIZE * sizeof(struct kValues));
	hipMemcpy(*phiID, phiIH, numK * sizeof(struct kValues), hipMemcpyHostToDevice);
//    gettimeofday(&time1, &tz);
//    memoryTime += (time1.tv_sec - time0.tv_sec) * 1000000 + time1.tv_usec - time0.tv_usec;

	//calculate phiMag
//	gettimeofday(&time0, &tz);
	ComputePhiMagGPU<<<dim_grid, dim_block>>> (*kValsD, *phiRD, *phiID);
	hipDeviceSynchronize();
//    gettimeofday(&time1, &tz);
//    kernelTime += (time1.tv_sec - time0.tv_sec) * 1000000 + time1.tv_usec - time0.tv_usec;

    //launch kernel
	//multithreading could be used, but it's not necessary. Even 32*32*32 input(numK=3072) would occupy all threads (2560 for RTX2070) simultaneously, which
	//use around 2s of CPU. Multithreading would help if there are small inputs, but why not just do it on CPU?
	//multithreading will decrease 32x32x32 performance by half
	for (int indexX = 0; indexX < numX; indexX++) {

		//allocate result space. per indexX
		float* globalqrD;
		float* globalqiD;

//        gettimeofday(&time0, &tz);
		hipMalloc(&globalqrD, dim_grid.x * sizeof(float));
		hipMalloc(&globalqiD, dim_grid.x * sizeof(float));
//        gettimeofday(&time1, &tz);
//        memoryTime += (time1.tv_sec - time0.tv_sec) * 1000000 + time1.tv_usec - time0.tv_usec;

//        gettimeofday(&time0, &tz);
		ComputeQGPU<<<dim_grid, dim_block>>>(globalqrD, globalqiD, *kValsD, xH[indexX], yH[indexX], zH[indexX]);
        hipDeviceSynchronize();
//        gettimeofday(&time1, &tz);
//        kernelTime += (time1.tv_sec - time0.tv_sec) * 1000000 + time1.tv_usec - time0.tv_usec;

		//reduction
		int currentDataNum = dim_grid.x;
		int interval = 1;
		dim3 dim_grid_reduction, dim_block_reduction;
		while (currentDataNum != 1) {
			dim_grid_reduction.x = currentDataNum / REDUCTION_BLOCK_SIZE + (currentDataNum % REDUCTION_BLOCK_SIZE == 0 ? 0 : 1);
			dim_grid_reduction.y = 1;
			dim_grid_reduction.z = 1;
			dim_block_reduction.x = REDUCTION_BLOCK_SIZE;
			dim_block_reduction.y = 1;
			dim_block_reduction.z = 1;
//            gettimeofday(&time0, &tz);
			ImprovedReductionKernel<<<dim_grid_reduction, dim_block_reduction>>>(globalqrD, interval, currentDataNum);
			ImprovedReductionKernel<<<dim_grid_reduction, dim_block_reduction>>>(globalqiD, interval, currentDataNum);
			hipDeviceSynchronize();
//            gettimeofday(&time1, &tz);
//            kernelTime += (time1.tv_sec - time0.tv_sec) * 1000000 + time1.tv_usec - time0.tv_usec;
			interval *= REDUCTION_BLOCK_SIZE;
			currentDataNum = currentDataNum / REDUCTION_BLOCK_SIZE + (currentDataNum % REDUCTION_BLOCK_SIZE == 0 ? 0 : 1);
		}

//        gettimeofday(&time0, &tz);
		hipMemcpy(&(QrH[indexX]), globalqrD, sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(&(QiH[indexX]), globalqiD, sizeof(float), hipMemcpyDeviceToHost);
//        gettimeofday(&time1, &tz);
//        memoryTime += (time1.tv_sec - time0.tv_sec) * 1000000 + time1.tv_usec - time0.tv_usec;
	}

//    printf("kernel: %ld us\n", kernelTime);
//    printf("IO: %ld us\n", memoryTime);


}
